#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void count_pi(iterations)

{
	
	// initialise thread ID
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

 	// initialise shared array
        __shared__ int counter[256];
        counter[threadIdx.x] = 0;
 



	// initialise RNG
	hiprand_init(45287643, index, 0, &state[index]);

	// initialise the counter
	counter[threadId.x] = 0;

	// compute random values and increment counter
	for (int i = 0; i < iterations; i++)
	{

		float x = hiprand_uniform();
		float y = hiprand_uniform();
		counter[threadIdx.x] += 1 - int(x*x + y*y);

	}
	
	// reduction
	int i = blockDim.x/2;
	while (i!=0)
	{
		if (threadIdx.x < i)
		{
		counter[threadIdx.x] += cache[threadIdx.x + i];
		}
	i /= 2;
	__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		atomicAdd(count,counter[0]);
	}
}

int main() {

	int N = 100000000;
	int area = 0;

	count_pi<<<10,10>>>(N);
	
	printf("\nPi:	%f\n",(4*counter[0])/(float)N);
	return 0;
}






/*
int main() {
    
    int N=10000000;
    int area=0;

    for(int i=0; i<N; i++) {
        float x = ((float)rand())/RAND_MAX;
        float y = ((float)rand())/RAND_MAX;

        if(x*x + y*y <= 1.0f) area++;
    }
    printf("\nPi:\t%f\n", (4.0*area)/(float)N);

    return(0);
}
*/


