
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main() {
	
    int N=10000000;
    int area=0;

    for(int i=0; i<N; i++) {
        float x = ((float)rand())/RAND_MAX;
        float y = ((float)rand())/RAND_MAX;
        if(x*x + y*y <= 1.0f) area++;
    }
    printf("\nPi:\t%f\n", (4.0*area)/(float)N);

    return(0);
}