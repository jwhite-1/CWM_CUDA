#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stlib.h>



// reduction from lectures
__global__ void reduction(float *d_input, float *d_output)
{
	// allocate memory
	__shared__ float smem_array[NUM_ELS];

	int tid = threadIdx.x + blockDim.x*blockIdx.x;

	// each thread loads data into shared memory

	smem_array[tid] = d_input[tid];
	__syncthreads();

	// perform binary tree reduction

	for (int d = blockDim.x/2; d>0; d /= 2)
	{
	__syncthreads();
	if (tid<d) smem_array[tid] += smem_array[tid+d];
	}
	
	// first thread puts result into global memory
	
	if (tid==0) d_output[0] = smem_array[0];

}

// kernel to calculate pi
__global__ void pi_cuda(int iterations)
{
/*
Pseudocode:
	-Initialise RNGs per thread
	-Generate N pairs of random x,y per thread
		-Is x*x + y*y < 1? if so tick up counter
	-Total up number of counts across all threads and multiply
	-Divide by total number of trials to estimate PI
*/


	// initialise thread ID
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int counter = 0


	// 1. Create a new generator of the desired type (see Generator Types ) with hiprandCreateGenerator().
	hiprandCreateGenerator(HIPRAND_RNG_PSEUDO_XORWOW, seed, offset


	// 2. Set the generator options (see Generator Options); for example, use hiprandSetPseudoRandomGeneratorSeed() to set the seed.



	// 3. Allocate memory on the device with hipMalloc().



	// 4. Generate random numbers with hiprandGenerate() or another generation function.
	int i;
	float x_coor, y_coor;
	for (i=0; i<=iterations; i++) 
		{
		x_coor = hiprandGenerate();
		y_coor = hiprandGenerate();

		if (x_coor*x_coor + y_coor*y_coor) <= 1
			{
			counter++;		
			}
		}


	// 5. Use the results.



	// 6. If desired, generate more random numbers with more calls to hiprandGenerate().



	// 7. Clean up with hiprandDestroyGenerator(). 



atomicAdd()
}


int main() {

// run kernel to calculate many values with which to guess pi
pi_cuda<<<5,10>>>(iterations);


return(0);
}

