#include "hip/hip_runtime.h"
/*
modified from

https://bitbucket.org/jsandham/algorithms_in_cuda/src/master/monti_carlo_pi/

https://gist.github.com/akiross/17e722c5bea92bd2c310324eac643df6




*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <math.h>

__global__ void setup_kernel(hiprandState *state)
{
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	hiprand_init(134872847, index, 0, &state[index]);
}



__global__ void count_pi(hiprandState *state,int *count,int iterations)
{
	
	// initialise thread ID
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

 	// initialise shared array
        __shared__ int counter[256];
        counter[threadIdx.x] = 0;
	__syncthreads();



	// initialise RNG
	hiprand_init(45287643, index, 0, &state[tid]);

	// initialise the counter
	counter[threadId.x] = 0;

	// compute random values and increment counter
	for (int i = 0; i < iterations; i++)
	{

		float x = hiprand_uniform(&state[tid]);
		float y = hiprand_uniform(&state[tid]);
		counter[threadIdx.x] += 1 - int(x*x + y*y);

	}
	
	// reduction
	int i = blockDim.x/2;
	while (i!=0)
	{
		if (threadIdx.x < i)
		{
		counter[threadIdx.x] += cache[threadIdx.x + i];
		}
	i /= 2;
	__syncthreads();
	}
	// sum the values without threads clashing
	if (threadIdx.x == 0)
	{
		atomicAdd(count,counter[0]);
	}
}

int main() {

	// initialise variables
	int n = 256*256;
	int m = 100000000;
	int *h_count;
	int *d_count;
	hiprandState *d_state;
	float pi;

	// allocate mem
	h_count = (int*)malloc(n*sizeof(int));
	hipMalloc((void**)&d_count, n*sizeof(int));
	hipMalloc((void**)&d_state, n*sizeof(hiprandState));
	hipMemset(d_count, 0, sizeof(int));

	// set up timing stuff
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start, 0);

	// set kernel
	dim3 gridSize = 256;
	dim3 blockSize = 256;

	setup_kernel<<<gridSize, blockSize>>>(d_state, d_count, m);

	// run simulation
	monte_pi<<<gridSize, blockSize>>(d_state,d_count,m);

	// copy results back to the host
	hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

	pi = *h_count*4.0/(n*m);
	printf("Value of pi calculated is: %f",pi);


	int area = 0;

	count_pi<<<10,10>>>(N);
	
	printf("\nPi:	%f\n",(4*counter[0])/(float)N);
	return 0;
}






/*
int main() {
    
    int N=10000000;
    int area=0;

    for(int i=0; i<N; i++) {
        float x = ((float)rand())/RAND_MAX;
        float y = ((float)rand())/RAND_MAX;

        if(x*x + y*y <= 1.0f) area++;
    }
    printf("\nPi:\t%f\n", (4.0*area)/(float)N);

    return(0);
}
*/


