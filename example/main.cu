#include "hip/hip_runtime.h"
/*
modified from

https://github.com/wesarmour/CWM-in-HPC-and-Scientific-Computing-2019

https://bitbucket.org/jsandham/algorithms_in_cuda/src/master/monti_carlo_pi/
*/

#include <stdio.h>
//#include <stlib.h>
#include <time.h>
//#include <random>
#include <hiprand.h>
#include <math.h>
#include "kernels.cuh"



int main()
{
	unsigned int n = 256*256;
	unsigned int m = 20000;
	int *h_count;
	int *d_count;
	hiprandState *d_state;
	float pi;


	// allocate memory
	h_count = (int*)malloc(n*sizeof(int));
	hipMalloc((void**)&d_count, n*sizeof(int));
	hipMalloc((void**)&d_state, n*sizeof(hiprandState));
	hipMemset(d_count, 0, sizeof(int));


	// set up timing stuff
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start, 0);


	// set kernel
	dim3 gridSize = 256;
	dim3 blockSize = 256;
	setup_kernel<<< gridSize, blockSize>>>(d_state);


	// monti carlo kernel
	monti_carlo_pi_kernel<<<gridSize, blockSize>>>(d_state, d_count, m);


	// copy results back to the host
	hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);


	// display results and timings for gpu
	pi = *h_count*4.0/(n*m);
	printf("Approximate pi calculated on GPU is: %f",pi);

/*
	//  serial verion
	clock_t cpu_start = clock();
	std::default_random_engine generator;
	std::uniform_real_distribution<float> distribution(0, 1.0);
	int count = 0;
	for(unsigned int i=0;i<n;i++){
		int temp = 0;
		while(temp < m){
			float x = distribution(generator);
			float y = distribution(generator);
			float r = x*x + y*y;

			if(r <= 1){
				count++;
			}
			temp++; 
		}
	}
	clock_t cpu_stop = clock();
	pi = 4.0*count/(n*m);
	std::cout<<"Approximate pi calculated on CPU is: "<<pi<<" and calculation took "<<1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC<<std::endl;


*/
	// delete memory
	free(h_count);
	hipFree(d_count);
	hipFree(d_state);
}

